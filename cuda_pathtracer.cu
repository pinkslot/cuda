#include "hip/hip_runtime.h"
/*
*  CUDA based triangle mesh path tracer using BVH acceleration by Sam lapere, 2016
*  BVH implementation based on real-time CUDA ray tracer by Thanassis Tsiodras, 
*  http://users.softlab.ntua.gr/~ttsiod/cudarenderer-BVH.html 
*  Interactive camera with depth of field based on CUDA path tracer code 
*  by Peter Kutz and Yining Karl Li, https://github.com/peterkutz/GPUPathTracer
*
*  This program is free software; you can redistribute it and/or modify
*  it under the terms of the GNU General Public License as published by
*  the Free Software Foundation; either version 2 of the License, or
*  (at your option) any later version.
*
*  This program is distributed in the hope that it will be useful,
*  but WITHOUT ANY WARRANTY; without even the implied warranty of
*  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*  GNU General Public License for more details.
*
*  You should have received a copy of the GNU General Public License
*  along with this program; if not, write to the Free Software
*  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA
*/
 
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>
#include <math_functions.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include ""
#include "cutil_math.h"
#include "C:\Program Files\NVIDIA Corporation\Installer2\CUDASamples_7.5.{075424A8-24ED-4D5A-B1EB-F0B5B2EDDCFB}\common\inc\GL\glew.h"
#include "C:\Program Files\NVIDIA Corporation\Installer2\CUDASamples_7.5.{075424A8-24ED-4D5A-B1EB-F0B5B2EDDCFB}\common\inc\GL\freeglut.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "cuda_pathtracer.h"

#define M_PI 3.1415926535897932384626422832795028841971f
#define TWO_PI 6.2831853071795864769252867665590057683943f
#define NUDGE_FACTOR     1e-3f  // epsilon
#define samps  1 // samples
#define BVH_STACK_SIZE 32
#define SCREEN_DIST (height*2)

int texturewidth = 0;
int textureheight = 0;
int total_number_of_triangles;

__device__ int depth = 0;


// Textures for vertices, triangles and BVH data
// (see CudaRender() below, as well as main() to see the data setup process)
texture<uint1, 1, hipReadModeElementType> g_triIdxListTexture;
texture<float2, 1, hipReadModeElementType> g_pCFBVHlimitsTexture;
texture<uint4, 1, hipReadModeElementType> g_pCFBVHindexesOrTrilistsTexture;
texture<float4, 1, hipReadModeElementType> g_trianglesTexture;

Vertex* cudaVertices;
float* cudaTriangleIntersectionData;
int* cudaTriIdxList = NULL;
float* cudaBVHlimits = NULL;
int* cudaBVHindexesOrTrilists = NULL;
Triangle* cudaTriangles = NULL;
Camera* cudaRendercam = NULL;


struct Ray {
	float3 orig;	// ray origin
	float3 dir;		// ray direction	
	__device__ Ray(float3 o_, float3 d_) : orig(o_), dir(d_) {}
	__device__ Ray(Vector3Df o_, Vector3Df d_) : 
		orig(make_float3(o_.x, o_.y, o_.z)), 
		dir(make_float3(d_.x, d_.y, d_.z)) {}
};

enum Refl_t { DIFF, METAL, SPEC, REFR, COAT };  // material types

struct Sphere {

	float rad, emit;				// radius 
	float3 pos;	// position, emission, color 
	//Refl_t refl;			// reflection type (DIFFuse, SPECular, REFRactive)

	__device__ float intersect(const Ray &r) const { // returns distance, 0 if nohit 

		// Ray/sphere intersection
		// Quadratic formula required to solve ax^2 + bx + c = 0 
		// Solution x = (-b +- sqrt(b*b - 4ac)) / 2a
		// Solve t^2*d.d + 2*t*(o-p).d + (o-p).(o-p)-R^2 = 0 

		float3 op = pos - r.orig;  // 
		float t, epsilon = 0.01f;
		float b = dot(op, r.dir);
		float disc = b*b - dot(op, op) + rad*rad; // discriminant
		if (disc<0) return 0; else disc = sqrtf(disc);
		return (t = b - disc)>epsilon ? t : ((t = b + disc)>epsilon ? t : 0);
	}

};

__device__ Sphere spheres[] = {

	// sun
	//{ 1e5f, .3f, { 1e5f + 700.f, 0.f, 0.f }},  // 37, 34, 30  X: links rechts Y: op neer
	// sky
	{ 20.f, .05f, { 0.f, 0.f, 0.f }},
	// ground
	//{ 100000.f, 0.f, { 0.0f, -100001.2, 0.f }},


	//{ .5f, 0.f, {0.f, 0.f, 0.f } },

	// mountains
	//{ 4e4, { 50.0f, -4e4 - 30, -3000 }, { 0, 0, 0 }, { 0.2f, 0.2f, 0.2f }, DIFF },
	// white Mirr
	// { 1.1, { 0, 0, -2 }, { 0, 0.0, 0 }, { 0.9f, .9f, 0.9f }, SPEC }
	// Glass
	//{ 0.3, { 0.0f, -0.4, 4 }, { .0, 0., .0 }, { 0.9f, 0.9f, 0.9f }, DIFF },
	// Glass2
	//{ 22, { 87.0f, 22, 24 }, { 0, 0, 0 }, { 0.9f, 0.9f, 0.9f }, SPEC },
};


// Create OpenGL BGR value for assignment in OpenGL VBO buffer
__device__ int getColor(Vector3Df& p)  // converts Vector3Df colour to int
{
	return (((unsigned)p.z) << 16) | (((unsigned)p.y) << 8) | (((unsigned)p.x));
}

// Helper function, that checks whether a ray intersects a bounding box (BVH node)
__device__ bool RayIntersectsBox(const Vector3Df& originInWorldSpace, const Vector3Df& rayInWorldSpace, int boxIdx)
{
	// set Tnear = - infinity, Tfar = infinity
	//
	// For each pair of planes P associated with X, Y, and Z do:
	//     (example using X planes)
	//     if direction Xd = 0 then the ray is parallel to the X planes, so
	//         if origin Xo is not between the slabs ( Xo < Xl or Xo > Xh) then
	//             return false
	//     else, if the ray is not parallel to the plane then
	//     begin
	//         compute the intersection distance of the planes
	//         T1 = (Xl - Xo) / Xd
	//         T2 = (Xh - Xo) / Xd
	//         If T1 > T2 swap (T1, T2) /* since T1 intersection with near plane */
	//         If T1 > Tnear set Tnear =T1 /* want largest Tnear */
	//         If T2 < Tfar set Tfar="T2" /* want smallest Tfar */
	//         If Tnear > Tfar box is missed so
	//             return false
	//         If Tfar < 0 box is behind ray
	//             return false
	//     end
	// end of for loop

	float Tnear, Tfar;
	Tnear = -FLT_MAX;
	Tfar = FLT_MAX;

	float2 limits;

// box intersection routine
#define CHECK_NEAR_AND_FAR_INTERSECTION(c)							    \
    if (rayInWorldSpace.##c == 0.f) {						    \
	if (originInWorldSpace.##c < limits.x) return false;					    \
	if (originInWorldSpace.##c > limits.y) return false;					    \
	} else {											    \
	float T1 = (limits.x - originInWorldSpace.##c)/rayInWorldSpace.##c;			    \
	float T2 = (limits.y - originInWorldSpace.##c)/rayInWorldSpace.##c;			    \
	if (T1>T2) { float tmp=T1; T1=T2; T2=tmp; }						    \
	if (T1 > Tnear) Tnear = T1;								    \
	if (T2 < Tfar)  Tfar = T2;								    \
	if (Tnear > Tfar)	return false;									    \
	if (Tfar < 0.f)	return false;									    \
	}

	limits = tex1Dfetch(g_pCFBVHlimitsTexture, 3 * boxIdx); // box.bottom._x/top._x placed in limits.x/limits.y
	//limits = make_float2(cudaBVHlimits[6 * boxIdx + 0], cudaBVHlimits[6 * boxIdx + 1]);
	CHECK_NEAR_AND_FAR_INTERSECTION(x)
	limits = tex1Dfetch(g_pCFBVHlimitsTexture, 3 * boxIdx + 1); // box.bottom._y/top._y placed in limits.x/limits.y
	//limits = make_float2(cudaBVHlimits[6 * boxIdx + 2], cudaBVHlimits[6 * boxIdx + 3]);
	CHECK_NEAR_AND_FAR_INTERSECTION(y)
	limits = tex1Dfetch(g_pCFBVHlimitsTexture, 3 * boxIdx + 2); // box.bottom._z/top._z placed in limits.x/limits.y
	//limits = make_float2(cudaBVHlimits[6 * boxIdx + 4], cudaBVHlimits[6 * boxIdx + 5]);
	CHECK_NEAR_AND_FAR_INTERSECTION(z)

	// If Box survived all above tests, return true with intersection point Tnear and exit point Tfar.
	return true;
}


//////////////////////////////////////////
//	BVH intersection routine	//
//	using CUDA texture memory	//
//////////////////////////////////////////

// there are 3 forms of the BVH: a "pure" BVH, a cache-friendly BVH (taking up less memory space than the pure BVH)
// and a "textured" BVH which stores its data in CUDA texture memory (which is cached). The last one is gives the 
// best performance and is used here.

__device__ bool BVH_IntersectTriangles(
	int* cudaBVHindexesOrTrilists, const Vector3Df& origin, const Vector3Df& ray, unsigned avoidSelf,
	int& pBestTriIdx, Vector3Df& pointHitInWorldSpace, float& hitdist,
	float* cudaBVHlimits, float* cudaTriangleIntersectionData, int* cudaTriIdxList, Vector3Df& boxnormal)
{
	// in the loop below, maintain the closest triangle and the point where we hit it:
	pBestTriIdx = -1;
	float bestTriDist;

	// start from infinity
	bestTriDist = FLT_MAX;

	// create a stack for each ray
	// the stack is just a fixed size array of indices to BVH nodes
	int stack[BVH_STACK_SIZE];
	
	int stackIdx = 0;
	stack[stackIdx++] = 0; 
	Vector3Df hitpoint;

	// while the stack is not empty
	while (stackIdx) {
		
		// pop a BVH node (or AABB, Axis Aligned Bounding Box) from the stack
		int boxIdx = stack[stackIdx - 1];
		//uint* pCurrent = &cudaBVHindexesOrTrilists[boxIdx]; 
		
		// decrement the stackindex
		stackIdx--;

		// fetch the data (indices to childnodes or index in triangle list + trianglecount) associated with this node
		uint4 data = tex1Dfetch(g_pCFBVHindexesOrTrilistsTexture, boxIdx);

		// texture memory BVH form...
		// determine if BVH node is an inner node or a leaf node by checking the highest bit (bitwise AND operation)
		// inner node if highest bit is 1, leaf node if 0

		if (!(data.x & 0x80000000)) {   // INNER NODE

			// if ray intersects inner node, push indices of left and right child nodes on the stack
			if (RayIntersectsBox(origin, ray, boxIdx)) {
				stack[stackIdx++] = data.y; // right child node index
				stack[stackIdx++] = data.z; // left child node index
				// return if stack size is exceeded
				if (stackIdx>BVH_STACK_SIZE)
				{
					return false; 
				}
			}
		}
		else { // LEAF NODE
			for (unsigned i = data.w; i < data.w + (data.x & 0x7fffffff); i++) {
				// fetch the index of the current triangle
				int idx = tex1Dfetch(g_triIdxListTexture, i).x;
				// check if triangle is the same as the one intersected by previous ray
				// to avoid self-reflections/refractions
				if (avoidSelf == idx)
					continue; 
				// fetch triangle center and normal from texture memory
				float4 center = tex1Dfetch(g_trianglesTexture, 5 * idx);
				float4 normal = tex1Dfetch(g_trianglesTexture, 5 * idx + 1);
				// use the pre-computed triangle intersection data: normal, d, e1/d1, e2/d2, e3/d3
				float k = dot(normal, ray);
				if (k == 0.0f)
					continue; // this triangle is parallel to the ray, ignore it.
				float s = (normal.w - dot(normal, origin)) / k;
				if (s <= 0.0f) // this triangle is "behind" the origin.
					continue;
				if (s <= NUDGE_FACTOR)  // epsilon
					continue;
				Vector3Df hit = ray * s;
				hit += origin;

				// ray triangle intersection
				// Is the intersection of the ray with the triangle's plane INSIDE the triangle?
				float4 ee1 = tex1Dfetch(g_trianglesTexture, 5 * idx + 2);
				float kt1 = dot(ee1, hit) - ee1.w; 
				if (kt1<0.0f) continue;
				float4 ee2 = tex1Dfetch(g_trianglesTexture, 5 * idx + 3);
				float kt2 = dot(ee2, hit) - ee2.w; 
				if (kt2<0.0f) continue;
				float4 ee3 = tex1Dfetch(g_trianglesTexture, 5 * idx + 4);
				float kt3 = dot(ee3, hit) - ee3.w; 
				if (kt3<0.0f) continue;
				// ray intersects triangle, "hit" is the world space coordinate of the intersection.
				{
					// is this intersection closer than all the others?
					float hitZ = distancesq(origin, hit);
					if (hitZ < bestTriDist) {
						// maintain the closest hit
						bestTriDist = hitZ;
						hitdist = sqrtf(bestTriDist);
						pBestTriIdx = idx;
						pointHitInWorldSpace = hit;
						// store barycentric coordinates (for texturing, not used for now)
					}
				}
			}
		}
	}
	
	return pBestTriIdx != -1;
}

template<class T>
__device__ void printv(T &arr, char mark = ' ') {
	printf("%f, %f, %f %c%c%c\n", arr.x, arr.y, arr.z, mark, mark, mark);
}

//////////////////////
// PATH TRACING
//////////////////////
__device__ Vector3Df path_trace(hiprandState *randstate, Vector3Df rayorig, Vector3Df raydir, int avoidSelf,
	Triangle *pTriangles, int* cudaBVHindexesOrTrilists, float* cudaBVHlimits, float* cudaTriangleIntersectionData, int* cudaTriIdxList)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	// colour mask
	Vector3Df mask = Vector3Df(1.0f, 1.0f, 1.0f);
	// accumulated colour
	for (int bounces = 0; bounces < 6 && mask.lengthsq() > 1e-2; bounces++){  // iteration up to 4 bounces (instead of recursion in CPU code)
		int sphere_id = -1, triangle_id = -1, pBestTriIdx = -1;
		int geomtype = -1;
		const Triangle *pBestTri = NULL;
		Vector3Df neworig, newdir;

		float d = 1e10;
		float scene_t = 1e10;


		float hitdistance = 1e20;
		Vector3Df n, nl; // normal, oriented 

		// intersect all triangles in the scene stored in BVH
		Vector3Df boxnormal = Vector3Df(0, 0, 0);

		BVH_IntersectTriangles(
			cudaBVHindexesOrTrilists, rayorig, raydir, avoidSelf,
			pBestTriIdx, neworig, hitdistance, cudaBVHlimits,
			cudaTriangleIntersectionData, cudaTriIdxList, boxnormal);

		// intersect all spheres in the scene
		int numspheres = sizeof(spheres) / sizeof(Sphere);
		for (int i = numspheres; i--;){  // for all spheres in scene
			// keep track of distance from origin to closest intersection point
			if ((d = spheres[i].intersect(Ray(rayorig, raydir))) && d < scene_t){ 
				scene_t = d; sphere_id = i; geomtype = 1; 
			}
		}
		// set avoidSelf to current triangle index to avoid intersection between this triangle and the next ray, 
		// so that we don't get self-shadow or self-reflection from this triangle...
		avoidSelf = pBestTriIdx;

		if (scene_t >= 1e10) {
			return 0.;
		}

		if (hitdistance < scene_t && hitdistance > 0.002) // EPSILON
		{
			scene_t = hitdistance;
			triangle_id = pBestTriIdx;
			geomtype = 2;
		}
		// SPHERES:
		if (geomtype == 1){

			Sphere &sphere = spheres[sphere_id]; // hit object with closest intersection
			Vector3Df w(0, 1, -.5);
			w.normalize();
			neworig = rayorig + raydir * scene_t;
			neworig.normalize();

			return mask *(exp(4 - 4.f *((w - neworig)).length()));
			/*
			n = neworig - sphere.pos;
			n.normalize();
			nl = dot(n, raydir) < 0 ? n : n * -1;
			// pick two random numbers
			float phi = 2 * M_PI * hiprand_uniform(randstate);
			float r2 = hiprand_uniform(randstate);
			float r2s = sqrtf(r2);

			// compute orthonormal coordinate frame uvw with hitpoint as origin 
			Vector3Df u = cross((fabs(nl.x) > .1 ? Vector3Df(0, 1, 0) : Vector3Df(1, 0, 0)), nl); u.normalize();
			Vector3Df v = cross(nl, u);

			// compute cosine weighted random ray direction on hemisphere 
			newdir = u*cosf(phi)*r2s + v*sinf(phi)*r2s + nl*sqrtf(1 - r2);
/*			if (sphere_id == 1 && dot(newdir, neworig) > 0) {
				printf("%f %f %f\n%f %f %f++\n%f\n", neworig.x, neworig.y, neworig.z, nl.x, nl.y, nl.z, dot(newdir, neworig));
			}
			newdir.normalize();
			*/
		}

		// TRIANGLES:5
		if (geomtype == 2){
			pBestTri = &pTriangles[triangle_id];
			// CHECK NORMAL BEFORE EDITING THIS LINE
			n = pBestTri->_normal*1;  // normal
			//n = Vector3Df(0,0,1);
			n.normalize();
	//			printf(dot(n, rayInWorldSpace) < 0 ? "" : "#");
			bool into = dot(n, raydir) < 0;
			nl = into ? n : n * -1;
			
			//Vector3Df colour = pBestTri->_colorf;
#define MU 29.f
#define LAMBDA Vector3Df(.8f, .6f, .2f)
			/*if (x == gridDim.x * blockDim.x / 2 && y == gridDim.y * blockDim.y / 2) {
				printf("%d", into);
			}*/

			if (!into && exp(-MU * hitdistance) < hiprand_uniform(randstate)) {
				// scattering

				float x1 = raydir.x, x2 = raydir.y, x3 = raydir.z;

				float indic = hiprand_uniform(randstate),
					phi = hiprand_uniform(randstate) * 2 * M_PI,
					sin_ind = (1 - indic * indic);

				Vector3Df rand_dir = Vector3Df(cos(phi) * sin_ind, sin(phi) * sin_ind, indic);
				if (abs(x3 - 1) > 1e-5) {
					float denom = sqrt(1 - x3);
					newdir.x = dot(Vector3Df(x1 * x3 / denom, -x2 / denom, x1), rand_dir);
					newdir.y = dot(Vector3Df(x2 * x3 / denom, x1 / denom, x2), rand_dir);
					newdir.z = dot(Vector3Df(-denom, 0, x3), rand_dir);
				}
				else newdir = rand_dir;
				neworig = rayorig - raydir * log((exp(-MU * hitdistance) - 1) * hiprand_uniform(randstate) + 1) / MU;
				mask *= LAMBDA;
			}
			else {
#define MEDIA_K 1.f  // Index of Refraction air
#define OBJ_K 1.3f  // Index of Refraction glass/water
				float k = into ? MEDIA_K / OBJ_K : OBJ_K / MEDIA_K;  // IOR ratio of refractive materials

				float ddn = dot(raydir, nl);
				float cos2t = 1.0f - k * k * (1.f - ddn*ddn);
				Vector3Df rdir = raydir - n * 2.0f * dot(n, raydir);

				if (cos2t < 0.0f) // total internal reflection 
				{
					newdir = rdir;
					// offset origin next path segment to prevent self intersection
				}
				else // cos2t > 0
				{
					// compute direction of transmission ray
					Vector3Df tdir = raydir * k - nl * (ddn * k + sqrtf(cos2t));
					tdir.normalize();

					float R0 = (OBJ_K - MEDIA_K)*(OBJ_K - MEDIA_K) / (OBJ_K + MEDIA_K)*(OBJ_K + MEDIA_K);
					float c = 1.f - (into ? -ddn : dot(tdir, n));
					float Re = R0 + (1.f - R0) * c * c * c * c * c;

					// randomly choose reflection or transmission ray
					newdir = hiprand_uniform(randstate) < Re ?
						raydir - n * 2.0f * dot(n, raydir) : tdir;
				}
			}
		}

		// set up origin and direction of next path segment
		
		raydir = newdir;
		raydir.normalize();
		rayorig = neworig + raydir * 1e-4;
	}

	return Vector3Df(0, 0, 0);
}
union Colour  // 4 bytes = 4 chars = 1 float
{
	float c;
	uchar4 components;
};

// the core path tracing kernel, 
// running in parallel for all pixels
__global__ void CoreLoopPathTracingKernel(Vector3Df* output, Vector3Df* accumbuffer, Triangle* pTriangles, Camera* cudaRendercam,
	int* cudaBVHindexesOrTrilists, float* cudaBVHlimits, float* cudaTriangleIntersectionData,
	int* cudaTriIdxList, unsigned int framenumber, unsigned int hashedframenumber)
{

	// assign a CUDA thread to every pixel by using the threadIndex
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	// global threadId, see richiesams blogspot
	int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	// create random number generator and initialise with hashed frame number, see RichieSams blogspot
	hiprandState randState; // state of the random number generator, to prevent repetition
	hiprand_init(hashedframenumber + threadId, 0, 0, &randState);

	Vector3Df finalcol; // final pixel colour  
	Vector3Df rendercampos = Vector3Df(cudaRendercam->position.x, cudaRendercam->position.y, cudaRendercam->position.z);


	int i = (height - y - 1)*width + x; // pixel index in buffer
	int pixelx = x; // pixel x-coordinate on screen
	int pixely = height - y - 1; // pixel y-coordintate on screen

	finalcol = Vector3Df(0.0f, 0.0f, 0.0f); // reset colour to zero for every pixel	
	for (int s = 0; s < samps; s++){

		// compute primary ray direction
		// use camera view of current frame (transformed on CPU side) to create local orthonormal basis
		Vector3Df rendercamview = Vector3Df(cudaRendercam->view.x, cudaRendercam->view.y, cudaRendercam->view.z); rendercamview.normalize(); // view is already supposed to be normalized, but normalize it explicitly just in case.
		Vector3Df rendercamup = Vector3Df(cudaRendercam->up.x, cudaRendercam->up.y, cudaRendercam->up.z); rendercamup.normalize();

		Vector3Df horizontalAxis = cross(rendercamview, rendercamup); horizontalAxis.normalize(); // Important to normalize!
		Vector3Df verticalAxis = cross(horizontalAxis, rendercamview); verticalAxis.normalize(); // verticalAxis is normalized by default, but normalize it explicitly just for good measure.

		Vector3Df middle = rendercampos + rendercamview;
		Vector3Df horizontal = horizontalAxis * tanf(cudaRendercam->fov.x * 0.5 * (M_PI / 180)); // Now treating FOV as the full FOV, not half, so I multiplied it by 0.5. I also normzlized A and B, so there's no need to divide by the length of A or B anymore. Also normalized view and removed lengthOfView. Also removed the cast to float.
		Vector3Df vertical = verticalAxis * tanf(-cudaRendercam->fov.y * 0.5 * (M_PI / 180)); // Now treating FOV as the full FOV, not half, so I multiplied it by 0.5. I also normzlized A and B, so there's no need to divide by the length of A or B anymore. Also normalized view and removed lengthOfView. Also removed the cast to float.

		// anti-aliasing
		// calculate center of current pixel and add random number in X and Y dimension
		// based on https://github.com/peterkutz/GPUPathTracer 
		float jitterValueX = hiprand_uniform(&randState) - 0.5;
		float jitterValueY = hiprand_uniform(&randState) - 0.5;
		float sx = (jitterValueX + pixelx) / (cudaRendercam->resolution.x - 1);
		float sy = (jitterValueY + pixely) / (cudaRendercam->resolution.y - 1);

		// compute pixel on screen
		Vector3Df pointOnPlaneOneUnitAwayFromEye = middle + ( horizontal * ((2 * sx) - 1)) + ( vertical * ((2 * sy) - 1));
		Vector3Df pointOnImagePlane = rendercampos + ((pointOnPlaneOneUnitAwayFromEye - rendercampos) * cudaRendercam->focalDistance); // Important for depth of field!		

		// calculation of depth of field / camera aperture 
		// based on https://github.com/peterkutz/GPUPathTracer 
		
		Vector3Df aperturePoint;

		if (cudaRendercam->apertureRadius > 0.00001) { // the small number is an epsilon value.
		
			// generate random numbers for sampling a point on the aperture
			float random1 = hiprand_uniform(&randState);
			float random2 = hiprand_uniform(&randState);

			// randomly pick a point on the circular aperture
			float angle = TWO_PI * random1;
			float distance = cudaRendercam->apertureRadius * sqrtf(random2);
			float apertureX = cos(angle) * distance;
			float apertureY = sin(angle) * distance;

			aperturePoint = rendercampos + (horizontalAxis * apertureX) + (verticalAxis * apertureY);
		}
		else { // zero aperture
			aperturePoint = rendercampos;
		}

		// calculate ray direction of next ray in path
		Vector3Df apertureToImagePlane = pointOnImagePlane - aperturePoint; 
		apertureToImagePlane.normalize(); // ray direction, needs to be normalised
		Vector3Df rayInWorldSpace = apertureToImagePlane;
		// in theory, this should not be required
		rayInWorldSpace.normalize();

		// origin of next ray in path
		Vector3Df originInWorldSpace = aperturePoint;

		finalcol += path_trace(&randState, originInWorldSpace, rayInWorldSpace, -1, pTriangles, 
			cudaBVHindexesOrTrilists, cudaBVHlimits, cudaTriangleIntersectionData, cudaTriIdxList) * (1.0f/samps);
	}       

	// add pixel colour to accumulation buffer (accumulates all samples) 
	accumbuffer[i] += finalcol;
	// averaged colour: divide colour by the number of calculated frames so far
	Vector3Df tempcol = accumbuffer[i] / framenumber;

	Colour fcolour;
	Vector3Df colour = Vector3Df(clamp(tempcol.x, 0.0f, 1.0f), clamp(tempcol.y, 0.0f, 1.0f), clamp(tempcol.z, 0.0f, 1.0f));
	// convert from 96-bit to 24-bit colour + perform gamma correction
	fcolour.components = make_uchar4((unsigned char)(powf(colour.x, 1 / 2.2f) * 255), (unsigned char)(powf(colour.y, 1 / 2.2f) * 255), (unsigned char)(powf(colour.z, 1 / 2.2f) * 255), 1);
	// store pixel coordinates and pixelcolour in OpenGL readable outputbuffer
	output[i] = Vector3Df(x, y, fcolour.c);

}

bool g_bFirstTime = true;

// the gateway to CUDA, called from C++ (in void disp() in main.cpp)
void cudarender(Vector3Df* dptr, Vector3Df* accumulatebuffer, Triangle* cudaTriangles, int* cudaBVHindexesOrTrilists,
	float* cudaBVHlimits, float* cudaTriangleIntersectionData, int* cudaTriIdxList, 
	unsigned framenumber, unsigned hashedframes, Camera* cudaRendercam){

	if (g_bFirstTime) {
		// if this is the first time cudarender() is called,
		// bind the scene data to CUDA textures!
		g_bFirstTime = false;

		printf("g_triIndexListNo: %d\n", g_triIndexListNo);
		printf("g_pCFBVH_No: %d\n", g_pCFBVH_No);
		printf("g_verticesNo: %d\n", g_verticesNo);
		printf("g_trianglesNo: %d\n", g_trianglesNo);

		hipChannelFormatDesc channel1desc = hipCreateChannelDesc<uint1>();
		hipBindTexture(NULL, &g_triIdxListTexture, cudaTriIdxList, &channel1desc, g_triIndexListNo * sizeof(uint1));

		hipChannelFormatDesc channel2desc = hipCreateChannelDesc<float2>();
		hipBindTexture(NULL, &g_pCFBVHlimitsTexture, cudaBVHlimits, &channel2desc, g_pCFBVH_No * 6 * sizeof(float));

		hipChannelFormatDesc channel3desc = hipCreateChannelDesc<uint4>();
		hipBindTexture(NULL, &g_pCFBVHindexesOrTrilistsTexture, cudaBVHindexesOrTrilists, &channel3desc,
			g_pCFBVH_No * sizeof(uint4));

		//hipChannelFormatDesc channel4desc = hipCreateChannelDesc<float4>();
		//hipBindTexture(NULL, &g_verticesTexture, cudaPtrVertices, &channel4desc, g_verticesNo * 8 * sizeof(float));

		hipChannelFormatDesc channel5desc = hipCreateChannelDesc<float4>();
		hipBindTexture(NULL, &g_trianglesTexture, cudaTriangleIntersectionData, &channel5desc, g_trianglesNo * 20 * sizeof(float));
	}
	dim3 block(32, 32, 1);   // dim3 CUDA specific syntax, block and grid are required to schedule CUDA threads over streaming multiprocessors
	dim3 grid(width / block.x, height / block.y, 1);

	/*hipEvent_t     start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);*/

	CoreLoopPathTracingKernel << <grid, block >> >(dptr, accumulatebuffer, cudaTriangles, cudaRendercam, cudaBVHindexesOrTrilists,
		cudaBVHlimits, cudaTriangleIntersectionData, cudaTriIdxList, framenumber, hashedframes);
	// get stop time, and display the timing results
	/*hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float   elapsedTime;
	hipEventElapsedTime(&elapsedTime,
		start, stop);
	printf("Time to generate:  %3.1f ms\n", elapsedTime);*/
}
